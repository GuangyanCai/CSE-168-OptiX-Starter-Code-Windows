#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include <optixu/optixu_math_namespace.h>
#include "Payloads.h"
#include "Constants.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

// Camera info 

// TODO:: delcare camera varaibles here

rtDeclareVariable(int1, depth, , ); // recursion depth

RT_PROGRAM void generateRays() {


	// TODO: calculate the ray direction
	float3 eye = make_float3(0, 0, 0); // change this
	float3 dir = make_float3(0, 0, 1); // change this


	// Shoot a ray to compute the color of the current pixel
	Ray ray = make_Ray(eye, dir, BASIC_RAY, RAY_EPSILON, RT_DEFAULT_MAX);
	Payload payload;
	payload.depth = depth.x;
	rtTrace(root, ray, payload);

	// Write the result
	resultBuffer[launchIndex] = payload.result;
}

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );

RT_PROGRAM void miss() {
	// Set the result to be the background color if miss


	// TODO: change the color to be the background color


	payload.result = make_float3(1, 0, 0);
}

RT_PROGRAM void exception() {
	// Print any exception for debugging
	const unsigned int code = rtGetExceptionCode();
	rtPrintExceptionDetails();
}