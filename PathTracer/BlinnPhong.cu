#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include <optixu/optixu_math_namespace.h>
#include "Payloads.h"
#include "Geometries.h"
#include "Lights.h"
#include "Constants.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 

// TODO: declare attribute variables here 

RT_PROGRAM void closestHit() {
	float3 result = make_float3(1, 0, 0);


	// TODO: calculate the color based	on lighting


	payload.result = result;
}

rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );

RT_PROGRAM void anyHit() {
	

	// TODO: handle the situation that the shadow ray hit any objects


	rtTerminateRay();
}